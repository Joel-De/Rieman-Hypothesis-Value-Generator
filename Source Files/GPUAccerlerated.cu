#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"
#include <vector>
#include "CPUSingleThread.h"
#include <math.h>
#include <stdio.h>
#include "cuda/std/cmath"
#include <Windows.h>

// Cuda Device function to calculate Complex Exponents
__device__ hipDoubleComplex CuCmplxExp(double base, hipDoubleComplex exp)
{
    hipDoubleComplex res;
    float r, i;
    
    sincosf(exp.y * logf(base), &i, &r); // sincosf may show up as error, but compiles with nvcc
    res.x = pow(base, exp.x) * r;
    res.y = pow(base, exp.x) * i;
    return res;
}

// Equivelant function to calculate Complex Exponents for the host device
//__device__ hipDoubleComplex CuCmplxExp(double base, hipDoubleComplex exp)
//{
//    hipDoubleComplex res;
//    float r, i;
//
//    sincosf(exp.y * logf(base), &r, &i);
//    res.x = base * r;
//    res.y = base * i;
//    return res;
//}

__global__ void addKernel(hipDoubleComplex *CudaInput, hipDoubleComplex*CudaOutput, int accuracy, int size, int KernelNum)
{
    
    int i = blockDim.x * blockIdx.x + threadIdx.x + KernelNum;
    CudaOutput[i] = make_hipDoubleComplex(0, 0);
    if (i < size) {
        
        for (int c1 = 1; c1 < accuracy; c1++) {

            if ((c1 % 2) == 0) {
                CudaOutput[i] = hipCsub(CudaOutput[i], hipCdiv(make_hipDoubleComplex(1, 0), CuCmplxExp((double)c1, CudaInput[i])));
                
            }
            else {
                CudaOutput[i] = hipCadd(CudaOutput[i], hipCdiv(make_hipDoubleComplex(1, 0), CuCmplxExp((double)c1, CudaInput[i])));
            }
            
        }

    }
}

//Function to Generate Reiman Values using CUDA
std::vector <RiemanPair> GenerateValuesCuda(double ymin, double ymax, double step, double accuracy)
{
    std::vector<hipDoubleComplex> BaseValues;

    for (double c1 = ymin; c1 < ymax; c1 += step) {
        BaseValues.emplace_back(make_hipDoubleComplex(0.5, c1));
    }

    int size = BaseValues.size();
    
    hipDoubleComplex* InputValues = new hipDoubleComplex[BaseValues.size()];
    hipDoubleComplex* ReturnValues = new hipDoubleComplex[BaseValues.size()];
    InputValues = BaseValues.data();


    hipDoubleComplex*CudaInput = 0;
    hipDoubleComplex*CudaOutput = 0;

    

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);


    cudaStatus = hipMalloc((void**)&CudaInput, BaseValues.size() * sizeof(hipDoubleComplex));
    cudaStatus = hipMalloc((void**)&CudaOutput, BaseValues.size() * sizeof(hipDoubleComplex));


    cudaStatus = hipMemcpy(CudaInput, InputValues, BaseValues.size() * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(CudaOutput, ReturnValues, BaseValues.size() * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);



    for (int KernelNumber = 0; KernelNumber <= (ceil((float)BaseValues.size() / (32 * 32))); KernelNumber++) {
        addKernel << <32, 32 >> > (CudaInput, CudaOutput, accuracy, BaseValues.size(), (32*32)*KernelNumber);
        
        cudaStatus = hipGetLastError();
        if (hipSuccess != cudaStatus) {

            fprintf(stderr, "Cuda error: %s.\n",
            hipGetErrorString(cudaStatus));
            exit(EXIT_FAILURE);

        }
    }


    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(ReturnValues, CudaOutput, BaseValues.size() * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    std::vector<RiemanPair> ReturnData;

    for (int i = 0; i < BaseValues.size(); i++) {
        ReturnData.emplace_back(
            std::complex<double>(InputValues[i].x, InputValues[i].y),
            std::complex<double>(ReturnValues[i].x, ReturnValues[i].y)
        );

    }
    hipFree(CudaInput);
    hipFree(CudaOutput);
    
    return ReturnData;
}
